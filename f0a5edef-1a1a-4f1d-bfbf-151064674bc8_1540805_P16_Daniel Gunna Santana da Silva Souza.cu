#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <omp.h>
#define BLOCK_SIZE 1024

__global__ void distance(int * gax, int * gay, int * gac, float  * gdistances, int * gcategoria, int len, int x, int y) {

   unsigned int t = threadIdx.x;
   unsigned int b = blockIdx.x*blockDim.x;

          __shared__ float partialDis[BLOCK_SIZE];
          __shared__ float partialAC[BLOCK_SIZE];

        partialDis[threadIdx.x] = sqrtf(powf(x-gax[t + b],2) + powf(y-gay[t + b],2));
        partialAC[threadIdx.x] = gac[threadIdx.x + b];

        __syncthreads();
        int metade = 512;
        if((b + threadIdx.x) < len){
        for(unsigned int i = 0; i<metade;metade = metade/2 ){
                if(threadIdx.x < metade){
                        if(partialDis[threadIdx.x] > partialDis[threadIdx.x+metade]){
                                partialDis[threadIdx.x] = partialDis[threadIdx.x+metade];
                                partialAC[threadIdx.x] = partialAC[threadIdx.x+metade];


                        }
                        __syncthreads();
                }
        }
        }
        __syncthreads();
        if(t == 0){
                gdistances[blockIdx.x] = partialDis[0];
                gcategoria[blockIdx.x] = partialAC[0];
        }

}


int main(int argc, char ** argv) {

   FILE * fp;
  int x,y,t;
  unsigned int i;
  int *ax, *ay, *ac;
  int ox, oy, oc;
  float *distances;
  float sDistance;

  int *gax, *gay, *gac;
  float *gdistances;
  int * gcategoria;
  /* Reading inputs */

   fp = fopen ("input.txt", "r");
   fscanf(fp,"%d %d",&x,&y);
   fscanf(fp,"%d",&t);

   ax = (int*) malloc(t*sizeof(int));
   ay = (int*) malloc(t*sizeof(int));
   ac = (int*) malloc(t*sizeof(int));
   int  * teste  = (int*) malloc(t * sizeof(int));
   float * teste2 = (float*) malloc( (ceil(t/BLOCK_SIZE) * sizeof(float)));

   hipMalloc((void**) &gax, t * sizeof(int));
   hipMalloc((void**) &gay, t * sizeof(int));
   hipMalloc((void**) &gac, t * sizeof(int));
   hipMalloc((void**) &gdistances, t *  sizeof(float));
   hipMalloc((void**) &gcategoria, ceil(t/BLOCK_SIZE) * sizeof(int));

  for(i=0; i < t; i++)
    fscanf(fp,"%d %d %d",&ax[i],&ay[i],&ac[i]);

   fclose(fp);
   hipMemcpy(gax, ax, t, hipMemcpyHostToDevice);
   hipMemcpy(gay, ay, t, hipMemcpyHostToDevice);
   hipMemcpy(gac, ac, t, hipMemcpyHostToDevice);

  //@@ Define grid and block sizes  
   dim3 DimGrid(ceil(t/BLOCK_SIZE), 1, 1);
   dim3 DimBlock(BLOCK_SIZE, 1, 1);
   distance<<<DimGrid, DimBlock>>>(gax, gay, gac, gdistances, gcategoria, t, x, y);

   hipMemcpy(ax, gax, t, hipMemcpyDeviceToHost);
   hipMemcpy(ay, gay, t, hipMemcpyDeviceToHost);
   hipMemcpy(ac, gac, t, hipMemcpyDeviceToHost);
   hipMemcpy(teste, gdistances, ceil(t/BLOCK_SIZE), hipMemcpyDeviceToHost);
   hipMemcpy(teste2, gcategoria, ceil(t/BLOCK_SIZE), hipMemcpyDeviceToHost);
   hipFree(gax);
   hipFree(gay);
   hipFree(gac);
   hipFree(gdistances);
   hipFree(gcategoria);

  double start = omp_get_wtime();

  /* Calculating nearest neighbor */
  sDistance = FLT_MAX;
  distances = (float*) malloc(ceil(t/BLOCK_SIZE)*sizeof(float));

  for(i=0; i < ceil(t/BLOCK_SIZE); i++) {
    if (sDistance > teste[i]) {
         sDistance = distances[i];
         oc = teste2[i];
    }
  }

  double end = omp_get_wtime();
  printf("\nTime = %f",end-start);

  printf("\nCategory = %d\n",oc);



        return 0;
}
~                                                                                               
